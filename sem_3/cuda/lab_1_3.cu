#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#ifdef _DEBUG

#define CHECK_ERR(a) { err = hipGetLastError(); \
				 if(err != hipSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, hipGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif


void sum_vec_host(double* A, double* B, double* C, int N)
{
	for (int i = 0; i < N; i++)
	{
		C[i] = A[i] + B[i];
	}
}

__global__ void sum_vec_device(double* A, double* B, double* C, int N)
{
	unsigned int line_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (line_id < N)
	{
		C[line_id] = A[line_id] + B[line_id];
	}
}

int main()
{
	// ���������� ����������
	int N = 1000000;
	double* h_A, * h_B, * h_C;
	double* d_A, * d_B, * d_C;
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	// ��������� ������ ��� ������� �������
	h_A = (double*)malloc(N * sizeof(double));
	h_B = (double*)malloc(N * sizeof(double));
	h_C = (double*)malloc(N * sizeof(double));

	hipMalloc((void**)&d_A, N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B, N * sizeof(double)); CHECK_ERR("malloc_2\n");
	hipMalloc((void**)&d_C, N * sizeof(double)); CHECK_ERR("malloc_3\n");
	hipEventCreate(&event1);
	hipEventCreate(&event2);

	// ������������� ���������� �������
	for (int i = 0; i < N; i++)
	{
		h_A[i] = 1.0;
		h_B[i] = 2.0;
		h_C[i] = 0.0;
		//if (i < 10) printf(" %le %le %le\n", h_A[i], h_B[i], h_C[i]);
	}

	hipEventRecord(event1, 0);
	hipMemcpy(d_A, h_A, N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_1\n");
	hipMemcpy(d_B, h_B, N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_2\n");
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&time_device_copy, event1, event2);
	printf("Time copy from device %le msec\n", time_device_copy);

	// kernel
	int threads = 32;
	int blocks = N / threads + 1;
	hipEventRecord(event1, 0);
	sum_vec_device << <blocks, threads >> > (d_A, d_B, d_C, N); CHECK_ERR("kernel_1\n");		// fine
	//sum_vec_device <<<threads, blocks>>> (d_A, d_B, d_C, N); CHECK_ERR("kernel_2\n");		// error
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2); // ����� ����������� �������_2, ���� ��������������
	hipEventElapsedTime(&time_device, event1, event2);
	printf("Time work on device %le msec\n", time_device);

	hipMemcpy(d_C, h_C, N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_3\n");


	// �������� ��������
	t1 = clock();
	sum_vec_host(h_A, h_B, h_C, N);
	t2 = clock();
	printf("Time work on host %le msec\n", ((double)(t2 - t1) / CLOCKS_PER_SEC) * 1000);

	// ������������ ������
	free(h_A); free(h_B); free(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

//Ctrl + s (����������)
//Ctrl + F7
//Ctrl + F5
