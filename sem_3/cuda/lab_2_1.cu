
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"


#ifdef _DEBUG

#define CHECK_ERR(a) { err = cudaGetLastError(); \
				 if(err != cudaSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, cudaGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif


void mult_mat_host(double* A, double* B, double* C, int N)
{
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			for (int k = 0; k < N; k++)
			{
				C[i * N + j] += A[i * N + k] * B[k * N + j];
			}
}

__global__ void mult_mat_device(double* A, double* B, double* C, int N)
{
	unsigned int line_id = threadIdx.x + blockIdx.x * blockDim.x;
	int i = line_id / N;
	int j = line_id % N;
	if (i < N && j < N)
	{
		for (int k = 0; k < N; k++)
		{
			C[i * N + j] += A[i * N + k] * B[k * N + j];
		}
	}
}

int main()
{
	// ���������� ����������
	int N = 1024;
	double* h_A, * h_B, * h_C;
	double* d_A, * d_B, * d_C;
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	// ��������� ������ ��� ������� �������
	h_A = (double*)malloc(N * N * sizeof(double));
	h_B = (double*)malloc(N * N * sizeof(double));
	h_C = (double*)malloc(N * N * sizeof(double));

	hipMalloc((void**)&d_A, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B, N * N * sizeof(double)); CHECK_ERR("malloc_2\n");
	hipMalloc((void**)&d_C, N * N * sizeof(double)); CHECK_ERR("malloc_3\n");
	hipEventCreate(&event1);
	hipEventCreate(&event2);

	// ������������� ���������� �������
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			h_A[i * N + j] = 1.0;
			h_B[i * N + j] = 2.0;
			h_C[i * N + j] = 0.0;
		}

	hipEventRecord(event1, 0);
	hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_1\n");
	hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_2\n");
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&time_device_copy, event1, event2);
	printf("Time copy from device %le msec\n", time_device_copy);

	// kernel
	int threads = 32;
	int blocks = (N * N) / threads + 1;
	hipEventRecord(event1, 0);
	mult_mat_device <<<blocks, threads >>> (d_A, d_B, d_C, N); CHECK_ERR("kernel_1\n");		// fine
	//sum_vec_device <<<threads, blocks>>> (d_A, d_B, d_C, N); CHECK_ERR("kernel_2\n");		// error
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2); // ����� ����������� �������_2, ���� ��������������
	hipEventElapsedTime(&time_device, event1, event2);
	printf("Time work on device %le msec\n", time_device);

	hipMemcpy(h_C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR("memcpy_3\n");


	// �������� ��������
	t1 = clock();
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			h_C[i * N + j] = 0.0;
		}
	mult_mat_host(h_A, h_B, h_C, N);
	t2 = clock();
	printf("Time work on host %le msec\n", ((double)(t2 - t1) / CLOCKS_PER_SEC) * 1000);

	// ������������ ������
	free(h_A); free(h_B); free(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

//Ctrl + s (����������)
//Ctrl + F7
//Ctrl + F5
