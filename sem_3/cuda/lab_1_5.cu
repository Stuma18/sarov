
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"


#ifdef _DEBUG

#define CHECK_ERR(a) { err = cudaGetLastError(); \
				 if(err != cudaSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, cudaGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif


void sum_vec_host(double* A, double* B, double* C, int N)
{
	for (int i = 0; i < N; i++)
	{
		C[i] = A[i] + B[i];
	}
}

__global__ void sum_vec_device(double* A, double* B, double* C, int N)
{
	unsigned int line_id = threadIdx.x + blockIdx.x * blockDim.x;
	if (line_id < N)
	{
		C[line_id] = A[line_id] + B[line_id];
	}
}

int main()
{
	// ���������� ����������
	int N = 1000000;
	double* h_A, * h_B, * h_C;
	double* d_A, * d_B, * d_C;
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	hipStream_t stream[2];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	// ��������� ������ ��� ������� �������
	hipHostMalloc((void**)&h_A, N * sizeof(double), hipHostMallocDefault); CHECK_ERR("malloc_1\n");
	hipHostMalloc((void**)&h_B, N * sizeof(double), hipHostMallocDefault); CHECK_ERR("malloc_1\n");
	hipHostMalloc((void**)&h_C, N * sizeof(double), hipHostMallocDefault); CHECK_ERR("malloc_1\n");

	hipMalloc((void**)&d_A, N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B, N * sizeof(double)); CHECK_ERR("malloc_2\n");
	hipMalloc((void**)&d_C, N * sizeof(double)); CHECK_ERR("malloc_3\n");
	hipEventCreate(&event1);
	hipEventCreate(&event2);

	// ������������� ���������� �������
	for (int i = 0; i < N; i++)
	{
		h_A[i] = 1.0;
		h_B[i] = 2.0;
		h_C[i] = 0.0;
	}

	hipEventRecord(event1, 0);
	for (int i = 0; i < 2; i++)
	{
		hipMemcpyAsync(d_A + i * N / 2, h_A + i * N / 2, (N / 2) * sizeof(double), hipMemcpyHostToDevice, stream[i]); CHECK_ERR("memcpy_1\n");
		hipMemcpyAsync(d_B + i * N / 2, h_B + i * N / 2, (N / 2) * sizeof(double), hipMemcpyHostToDevice, stream[i]); CHECK_ERR("memcpy_2\n");
	}

	// kernel
	int threads = 32;
	int blocks = N / (2 * threads) + 1;
	for (int i = 0; i < 2; i++)
	{
		sum_vec_device <<<blocks, threads, 0, stream[i]>>> (d_A + i * N / 2, d_B + i * N / 2, d_C + i * N / 2, N / 2); CHECK_ERR("kernel_1\n");
	}

	for (int i = 0; i < 2; i++)
	{
		hipMemcpyAsync(d_C + i * N / 2, h_C + i * N / 2, (N / 2) * sizeof(double), hipMemcpyHostToDevice, stream[i]); CHECK_ERR("memcpy_3\n");
	}

	hipEventRecord(event2, 0);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&time_device_copy, event1, event2);
	printf("Time work on device %le msec\n", time_device_copy);
	
	// �������� ��������
	t1 = clock();
	sum_vec_host(h_A, h_B, h_C, N);
	t2 = clock();
	printf("Time work on host %le msec\n", ((double)(t2 - t1) / CLOCKS_PER_SEC) * 1000);

	// ������������ ������
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	for (int i = 0; i < 2; i++)
	{
		hipStreamDestroy(stream[i]);
	}
}

//Ctrl + s (����������)
//Ctrl + F7
//Ctrl + F5
