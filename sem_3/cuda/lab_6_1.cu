#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#define L 5					// ������� ��������
#define sigma 1				// ���������


#ifdef _DEBUG

#define CHECK_ERR(a) { err = hipGetLastError(); \
				 if(err != hipSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, hipGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif


void host(int N)
{
	double x, l, csi;
	int t = 0;

	while (N > 0)
	{
		x = 0;
		csi = (double)rand() / (double)(RAND_MAX + 1);
		l = -log(csi) / sigma;
		x += l;
		if (x > L)
		{
			t++;
		}
		N--;
	}
	printf("%d\n", t);
}


__global__ void device(int N, int* Nplus)
{
	unsigned int seed = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState s;
	hiprand_init(seed, 0, 0, &s);
	float csi = hiprand_uniform(&s);
	double x, l;
	int t = 0;

	x = 0;
	l = -log(csi) / sigma;
	x += l;
	if (x > L)
	{
		atomicAdd(Nplus, 1);
	}
}



int main()
{
	int N = 1000000;				// ����� ���������
	srand(time(NULL));
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	t1 = clock();
	host(N);
	t2 = clock();
	printf("Time work on host %lf msec\n", (double)(t2 - t1) / CLOCKS_PER_SEC * 1000);

	hipEventCreate(&event1);
	hipEventCreate(&event2);

	int* Nplus;
	hipMalloc(&Nplus, sizeof(int));

	int threads = 32;
	int blocks = N / threads + 1;

	hipEventRecord(event1, 0);
	device << <blocks, threads >> > (N, Nplus); CHECK_ERR("kernel_1\n");
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&time_device, event1, event2);

	hipMemcpy(&Nplus, Nplus, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", Nplus);
	printf("Time work on device %lf msec\n", time_device); CHECK_ERR("kernel_15\n");

	hipFree(Nplus);

	int N_a = exp(-L) * N;
	printf("%d\n", N_a);

	return 0;
}
