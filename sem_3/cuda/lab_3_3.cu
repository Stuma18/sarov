#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#ifdef _DEBUG

#define CHECK_ERR(a) { err = hipGetLastError(); \
				 if(err != hipSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, hipGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif

void fil_SoA_host(double* h_A_r, double* h_A_g, double* h_A_b, double* h_B_r, double* h_B_g, double* h_B_b, int N)
{
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
			{
				h_B_r[i * N + j] = h_A_r[i * N + j];
				h_B_g[i * N + j] = h_A_g[i * N + j];
				h_B_b[i * N + j] = h_A_b[i * N + j];
			}
			else
			{
				for (int k = i - 1; k < i + 2; k++)
					for (int z = j - 1; z < j + 2; z++)
					{
						h_B_r[k * N + z] += h_A_r[k * N + z];
						h_B_g[k * N + z] += h_A_g[k * N + z];
						h_B_b[k * N + z] += h_A_b[k * N + z];
					}
				h_B_r[i * N + j] = h_B_r[i * N + j] / 9;
				h_B_g[i * N + j] = h_B_g[i * N + j] / 9;
				h_B_b[i * N + j] = h_B_b[i * N + j] / 9;
			}
}

__global__ void fil_SoA_device(double* d_A_r, double* d_B_r, int N)
{
	unsigned int line_id = threadIdx.x + blockIdx.x * blockDim.x;
	int i = line_id / N;
	int j = line_id % N;
	if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
	{
		d_B_r[i * N + j] = d_A_r[i * N + j];
		//d_B_g[i * N + j] = d_A_g[i * N + j];
		//d_B_b[i * N + j] = d_A_b[i * N + j];
	}
	else
	{
		for (int k = i - 1; k < i + 2; k++)
			for (int z = j - 1; z < j + 2; z++)
			{
				d_B_r[k * N + z] += d_A_r[k * N + z];
				//d_B_g[k * N + z] += d_A_g[k * N + z];
				//d_B_b[k * N + z] += d_A_b[k * N + z];
			}
		d_B_r[i * N + j] = d_B_r[i * N + j] / 9;
		//d_B_g[i * N + j] = d_B_g[i * N + j] / 9;
		//d_B_b[i * N + j] = d_B_b[i * N + j] / 9;
	}
}

int main()
{
	// ���������� ����������
	int N = 1024;
	double* h_A_r, * h_A_g, * h_A_b, * h_B_r, * h_B_g, * h_B_b;
	double* d_A_r, * d_A_g, * d_A_b, * d_B_r, * d_B_g, * d_B_b;
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	hipStream_t stream[3];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);
	hipStreamCreate(&stream[2]);

	// ��������� ������ ��� ������� �������
	h_A_r = (double*)malloc(N * N * sizeof(double));
	h_A_g = (double*)malloc(N * N * sizeof(double));
	h_A_b = (double*)malloc(N * N * sizeof(double));
	h_B_r = (double*)malloc(N * N * sizeof(double));
	h_B_g = (double*)malloc(N * N * sizeof(double));
	h_B_b = (double*)malloc(N * N * sizeof(double));

	hipMalloc((void**)&d_A_r, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_A_g, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_A_b, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_r, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_g, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_b, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipEventCreate(&event1);
	hipEventCreate(&event2);

	// ������������� ���������� �������
	srand(time(0));
	for (int i = 0; i < N * N; i++)
	{
		h_A_r[i] = rand() % 256;
		h_A_g[i] = rand() % 256;
		h_A_b[i] = rand() % 256;
		h_B_r[i] = 0;
		h_B_g[i] = 0;
		h_B_b[i] = 0;
	}

	hipEventRecord(event1, 0);
	
	hipMemcpyAsync(d_A_r, h_A_r, N * N * sizeof(double), hipMemcpyHostToDevice, stream[0]); CHECK_ERR("memcpy_1\n");
	hipMemcpyAsync(d_A_g, h_A_g, N * N * sizeof(double), hipMemcpyHostToDevice, stream[1]); CHECK_ERR("memcpy_2\n");
	hipMemcpyAsync(d_A_b, h_A_b, N * N * sizeof(double), hipMemcpyHostToDevice, stream[2]); CHECK_ERR("memcpy_3\n");
	

	//hipEventRecord(event2, 0);
	//hipEventSynchronize(event2);
	//hipEventElapsedTime(&time_device_copy, event1, event2);
	//printf("Time copy from device %le msec\n", time_device_copy);

	// kernel
	int threads = 32;
	int blocks = N / threads + 1;
	//hipEventRecord(event1, 0);
	
	fil_SoA_device << <blocks, threads, 0, stream[0] >> > (d_A_r, d_B_r, N); CHECK_ERR("kernel_1\n");		// fine
	fil_SoA_device << <blocks, threads, 0, stream[1] >> > (d_A_g, d_B_g, N); CHECK_ERR("kernel_1\n");
	fil_SoA_device << <blocks, threads, 0, stream[2] >> > (d_A_b, d_B_b, N); CHECK_ERR("kernel_1\n");

	/*
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2); // ����� ����������� �������_2, ���� ��������������
	hipEventElapsedTime(&time_device, event1, event2);
	printf("Time work on device %le msec\n", time_device);
	*/
	hipMemcpyAsync(h_B_r, d_B_r, N * N * sizeof(double), hipMemcpyDeviceToHost, stream[0]); CHECK_ERR("memcpy_4\n");
	hipMemcpyAsync(h_B_g, d_B_g, N * N * sizeof(double), hipMemcpyDeviceToHost, stream[1]); CHECK_ERR("memcpy_5\n");
	hipMemcpyAsync(h_B_b, d_B_b, N * N * sizeof(double), hipMemcpyDeviceToHost, stream[2]); CHECK_ERR("memcpy_6\n");

	hipEventRecord(event2, 0);
	hipEventSynchronize(event2); // ����� ����������� �������_2, ���� ��������������
	hipEventElapsedTime(&time_device, event1, event2);
	printf("Time work on device %le msec\n", time_device);


	// �������� ��������
	t1 = clock();
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			h_B_r[i * N + j] = 0.0;
			h_B_g[i * N + j] = 0.0;
			h_B_b[i * N + j] = 0.0;
		}
	fil_SoA_host(h_A_r, h_A_g, h_A_b, h_B_r, h_B_g, h_B_b, N);
	t2 = clock();
	printf("Time work on host %le msec\n", ((double)(t2 - t1) / CLOCKS_PER_SEC) * 1000);

	// ������������ ������
	free(h_A_r); free(h_A_g); free(h_A_b); free(h_B_r); free(h_B_g); free(h_B_b);
	hipFree(d_A_r); hipFree(d_A_g); hipFree(d_A_b); hipFree(d_B_r); hipFree(d_B_g); hipFree(d_B_b);

	for (int i = 0; i < 3; i++)
	{
		hipStreamDestroy(stream[i]);
	}
}
