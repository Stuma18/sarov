#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#ifdef _DEBUG

#define CHECK_ERR(a) { err = hipGetLastError(); \
				 if(err != hipSuccess) { printf(a); printf("%s(%d): %s \n", __FILE__, __LINE__, hipGetErrorString(err)); } }
#else
#define CHECK_ERR(a)
#endif


// ������� �������� �������� � ������� B:
void fil_SoA_host(double* h_A_r, double* h_A_g, double* h_A_b, double* h_B_r, double* h_B_g, double* h_B_b, int N)
{
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
			{
				h_B_r[i * N + j] = h_A_r[i * N + j];
				h_B_g[i * N + j] = h_A_g[i * N + j];
				h_B_b[i * N + j] = h_A_b[i * N + j];
			}
			else
			{
				for (int k = i - 1; k < i + 2; k++)
					for (int z = j - 1; z < j + 2; z++)
					{
						h_B_r[i * N + j] += h_A_r[k * N + z];
						h_B_g[i * N + j] += h_A_g[k * N + z];
						h_B_b[i * N + j] += h_A_b[k * N + z];
					}
				h_B_r[i * N + j] = h_B_r[i * N + j] / 9;
				h_B_g[i * N + j] = h_B_g[i * N + j] / 9;
				h_B_b[i * N + j] = h_B_b[i * N + j] / 9;
			}
}

__global__ void fil_SoA_device(double* d_A_r, double* d_A_g, double* d_A_b, double* d_B_r, double* d_B_g, double* d_B_b, int N)
{
	unsigned int line_id = threadIdx.x + blockIdx.x * blockDim.x;
	int i = line_id / N;
	int j = line_id % N;
	if (i < N && j < N)
	{
		if (i == 0 || i == N - 1 || j == 0 || j == N - 1)
		{
			d_B_r[i * N + j] = d_A_r[i * N + j];
			d_B_g[i * N + j] = d_A_g[i * N + j];
			d_B_b[i * N + j] = d_A_b[i * N + j];
		}
		else
		{
			for (int k = i - 1; k < i + 2; k++)
				for (int z = j - 1; z < j + 2; z++)
				{
					d_B_r[i * N + j] += d_A_r[k * N + z];
					d_B_g[i * N + j] += d_A_g[k * N + z];
					d_B_b[i * N + j] += d_A_b[k * N + z];
				}
			d_B_r[i * N + j] = d_B_r[i * N + j] / 9;
			d_B_g[i * N + j] = d_B_g[i * N + j] / 9;
			d_B_b[i * N + j] = d_B_b[i * N + j] / 9;
		}
	}
}

int main()
{
	// ���������� ����������
	int N = 1024;
	double* h_A_r, * h_A_g, * h_A_b, * h_B_r, * h_B_g, * h_B_b;
	double* d_A_r, * d_A_g, * d_A_b, * d_B_r, * d_B_g, * d_B_b;
	clock_t t1, t2;
	hipEvent_t event1, event2;
	float time_device = 0.0f;
	float time_device_copy = 0.0f;
	hipError_t err;

	// ��������� ������ ��� ������� �������
	h_A_r = (double*)malloc(N * N * sizeof(double));
	h_A_g = (double*)malloc(N * N * sizeof(double));
	h_A_b = (double*)malloc(N * N * sizeof(double));
	h_B_r = (double*)malloc(N * N * sizeof(double));
	h_B_g = (double*)malloc(N * N * sizeof(double));
	h_B_b = (double*)malloc(N * N * sizeof(double));

	hipMalloc((void**)&d_A_r, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_A_g, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_A_b, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_r, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_g, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipMalloc((void**)&d_B_b, N * N * sizeof(double)); CHECK_ERR("malloc_1\n");
	hipEventCreate(&event1);
	hipEventCreate(&event2);

	// ������������� ���������� �������
	srand(time(0));
	for (int i = 0; i < N * N; i++)
	{
		h_A_r[i] = rand() % 256;
		h_A_g[i] = rand() % 256;
		h_A_b[i] = rand() % 256;
		h_B_r[i] = 0;
		h_B_g[i] = 0;
		h_B_b[i] = 0;
	}
/*
		printf("h_A_r\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_A_r[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("h_A_g\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_A_g[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("h_A_b\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_A_b[i * N + j]);
			}
			printf("\n");
		}
*/	

	hipEventRecord(event1, 0);
	hipMemcpy(d_A_r, h_A_r, N * N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_1\n");
	hipMemcpy(d_A_g, h_A_g, N * N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_2\n");
	hipMemcpy(d_A_b, h_A_b, N * N * sizeof(double), hipMemcpyHostToDevice); CHECK_ERR("memcpy_3\n");
	hipEventRecord(event2, 0);
	hipEventSynchronize(event2);
	hipEventElapsedTime(&time_device_copy, event1, event2);
	printf("Time copy to device %le msec\n", time_device_copy);

	// kernel
	int threads = 32;
	int blocks = (N * N) / threads + 1;
	hipEventRecord(event1, 0);
	fil_SoA_device <<<blocks, threads >> > (d_A_r, d_A_g, d_A_b, d_B_r, d_B_g, d_B_b, N); CHECK_ERR("kernel_1\n");		// fine
	hipEventRecord(event2, 0); CHECK_ERR("kernel_12\n");
	hipEventSynchronize(event2); CHECK_ERR("kernel_13\n"); // ����� ����������� �������_2, ���� ��������������
	hipEventElapsedTime(&time_device, event1, event2); CHECK_ERR("kernel_14\n");
	printf("Time work on device %le msec\n", time_device); CHECK_ERR("kernel_15\n");

	hipMemcpy(h_B_r, d_B_r, N * N * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR("memcpy_4\n");
	hipMemcpy(h_B_g, d_B_g, N * N * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR("memcpy_5\n");
	hipMemcpy(h_B_b, d_B_b, N * N * sizeof(double), hipMemcpyDeviceToHost); CHECK_ERR("memcpy_6\n");

	/*
		printf("d_B_r\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_r[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("d_B_g\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_g[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("d_B_b\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_b[i * N + j]);
			}
			printf("\n");
		}
	*/
	// ����������
	t1 = clock();
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			h_B_r[i * N + j] = 0.0;
			h_B_g[i * N + j] = 0.0;
			h_B_b[i * N + j] = 0.0;
		}
	fil_SoA_host(h_A_r, h_A_g, h_A_b, h_B_r, h_B_g, h_B_b, N);
	t2 = clock();
	printf("Time work on host %le msec\n", ((double)(t2 - t1) / CLOCKS_PER_SEC) * 1000);
/*
		printf("h_B_r\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_r[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("h_B_g\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_g[i * N + j]);
			}
			printf("\n");
		}
		printf("\n");
		printf("h_B_b\n");
		for (int j = 0; j < N; j++) {
			for (int i = 0; i < N; i++) {
				printf("%lf\t", h_B_b[i * N + j]);
			}
			printf("\n");
		}
*/
	// ������������ ������
	free(h_A_r); free(h_A_g); free(h_A_b); free(h_B_r); free(h_B_g); free(h_B_b);
	hipFree(d_A_r); hipFree(d_A_g); hipFree(d_A_b); hipFree(d_B_r); hipFree(d_B_g); hipFree(d_B_b);
}
